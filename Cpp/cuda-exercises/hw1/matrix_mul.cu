
#include <hip/hip_runtime.h>
#include <stdio.h>

// Timing library for measuring performance
#include <time.h>


// Macro for error checking after CUDA API calls
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


// Matrix dimensions
const int DSIZE = 4096;
const int block_size = 16;  // Maximum threads per block dimension is 1024 (total)
const float A_val = 1.0f;   // Constant value for matrix A
const float B_val = 2.0f;   // Constant value for matrix B

// Matrix multiply (naive) kernel: C = A * B
__global__ void mmul(const float *A, const float *B, float *C, int ds) {
  
  // Calculate thread's global position within the grid
  int idx = threadIdx.x+blockDim.x*blockIdx.x; // Create thread x index - col
  int idy = threadIdx.y+blockDim.y*blockIdx.y; // Create thread y index - row

  // Check if thread is within matrix bounds
  if ((idx < ds) && (idy < ds)){
    float temp = 0;
    for (int i = 0; i < ds; i++)
      temp += A[idy*ds+i] * B[i*ds+idx];  // Accumulate dot product of row and column
    C[idy*ds+idx] = temp;  // Store result in C
  }
}

int main(){

  float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

  // Timing variables
  clock_t t0, t1, t2;
  double t1sum=0.0;
  double t2sum=0.0;

  // Start timing initialization
  t0 = clock();

  // Allocate and initialize host memory for matrices A, B, and C
  h_A = new float[DSIZE*DSIZE];
  h_B = new float[DSIZE*DSIZE];
  h_C = new float[DSIZE*DSIZE];
  for (int i = 0; i < DSIZE*DSIZE; i++){
    h_A[i] = A_val;
    h_B[i] = B_val;
    h_C[i] = 0;  // Initialize output matrix C to 0
  }

  // End of initialization timing
  t1 = clock();
  t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
  printf("Init took %f seconds.  Begin compute\n", t1sum);

  // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));
  hipMalloc(&d_B, DSIZE*DSIZE*sizeof(float));
  hipMalloc(&d_C, DSIZE*DSIZE*sizeof(float));
  cudaCheckErrors("hipMalloc failure");
  hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");

  // Cuda processing sequence step 1 is complete

  // Launch kernel - Define the grid and block dimensions
  dim3 block(block_size, block_size);  // dim3 variable holds 3 dimensions
  dim3 grid((DSIZE+block.x-1)/block.x, (DSIZE+block.y-1)/block.y);  // Any extra elements that don’t fully fit into a block will still get their own block
  mmul<<<grid, block>>>(d_A, d_B, d_C, DSIZE);
  cudaCheckErrors("kernel launch failure");

  // Cuda processing sequence step 2 is complete

  // Copy result matrix C back to host
  hipMemcpy(h_C, d_C, DSIZE*DSIZE*sizeof(float), hipMemcpyDeviceToHost);

  // End of GPU computation timing
  t2 = clock();
  t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
  printf ("Done. Compute took %f seconds\n", t2sum);

  // Cuda processing sequence step 3 is complete

  // Verify results on host
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
  for (int i = 0; i < DSIZE*DSIZE; i++) if (h_C[i] != A_val*B_val*DSIZE) {printf("mismatch at index %d, was: %f, should be: %f\n", i, h_C[i], A_val*B_val*DSIZE); return -1;}
  printf("Success!\n"); 

  return 0;
}
  
